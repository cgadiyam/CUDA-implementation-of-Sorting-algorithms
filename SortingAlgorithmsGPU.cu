#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>

__global__ void RankSortKernel(float* DataIn, float* DataOut, int* rank, int size)
{
	// Retrieve our coordinates in the block
	int tx = (blockIdx.x * 512) + threadIdx.x;
	rank[tx] = 0;
	if(tx < size)
	{
		for(int i=0;i<size;i++)
		{
			if(DataIn[tx]>=DataIn[i])
			{
				rank[tx]++;
			}
		}
		DataOut[(rank[tx]-1)] = DataIn[tx];
	}
}

__global__ void OddEvenSortKernel(float* Array, int size, bool Odd_Phase)
{
	int tx = (blockIdx.x * 512) + threadIdx.x;
	float temp;
	int index = 2*tx;
	if(tx < (size/2))
	{
		if(Odd_Phase == false)
		{
			if((index+1) < size)
			{
				if(Array[index]>Array[index+1])
				{
					temp = Array[index];
					Array[index] = Array[index+1];
					Array[index+1] = temp;
				}
			}
		}
		else
		{
			if((index+2) < size)
			{
				if(Array[index+1]>Array[index+2])
				{
					temp = Array[index+1];
					Array[index+1] = Array[index+2];
					Array[index+2] = temp;
				}
			}
		}
	}
}

bool RankSortGPU( float* InputArray, float* SortedArray, int size)
{
	int blocksize, gridsize;
	// Error return value
	hipError_t status;
	// Number of bytes
	int bytes = size * sizeof(float);
	// Pointers to the device arrays
	float *DataIn, *DataOut;
	int *rank;
	int bytes1 = size * sizeof(float);
	// Allocate memory on the device
	hipMalloc((void**) &DataIn, bytes);
	hipMalloc((void**) &DataOut, bytes);
	hipMalloc((void**) &rank, bytes1);
	// Copy the host input data to the device
	hipMemcpy(DataIn, InputArray, bytes, hipMemcpyHostToDevice);
	// Specify the size of the grid and the size of the block
	dim3 dimBlock(512, 1); 
	dim3 dimGrid((int)ceil((float)size/512), 1);
	// Launch the kernel on a size-by-size block of threads
	RankSortKernel<<<dimGrid, dimBlock>>>(DataIn, DataOut, rank, size);
	// Wait for completion
	hipDeviceSynchronize();
	// Check for errors
	status = hipGetLastError();
	if (status != hipSuccess) 
	{
		std::cout << "Kernel failed 1: " << hipGetErrorString(status) <<
		std::endl;
		hipFree(DataIn);
		hipFree(DataOut);
		return false;
	}
	// Retrieve the result matrix
	hipMemcpy(SortedArray, DataOut, bytes, hipMemcpyDeviceToHost);
	// Free device memory
	hipFree(DataIn);
	hipFree(DataOut);
	// Success
	return true;
}

bool OddEvenSortGPU( float* InputArray, float* SortedArray, int size)
{
	int blocksize, gridsize;
	// Error return value
	hipError_t status;
	// Number of bytes
	int bytes = size * sizeof(float);
	// Pointers to the device arrays
	float *Array;
	bool Odd_Phase;
	// Allocate memory on the device
	hipMalloc((void**) &Array, bytes);
	// Copy the host input data to the device
	hipMemcpy(Array, InputArray, bytes, hipMemcpyHostToDevice);
	int new_size = size/2;
	// Specify the size of the grid and the size of the block
	dim3 dimBlock(512, 1); 
	dim3 dimGrid((int)ceil((float)new_size/512), 1);
	for(int i=0;i<size;i++)
	{
		//even phase
		Odd_Phase = false;
		// Launch the kernel on a size-by-size block of threads
		OddEvenSortKernel<<<dimGrid, dimBlock>>>(Array, size, Odd_Phase);
		// Wait for completion
		hipDeviceSynchronize();
		// Check for errors
		status = hipGetLastError();
		if (status != hipSuccess) 
		{
			std::cout << "Kernel failed 2: " << hipGetErrorString(status) <<
			std::endl;
			hipFree(Array);
			return false;
		}
		//odd phase
		Odd_Phase = true;
		// Launch the kernel on a size-by-size block of threads
		OddEvenSortKernel<<<dimGrid, dimBlock>>>(Array, size, Odd_Phase);
		// Wait for completion
		hipDeviceSynchronize();
		// Check for errors
		status = hipGetLastError();
		if (status != hipSuccess) 
		{
			std::cout << "Kernel failed 3: " << hipGetErrorString(status) <<
			std::endl;
			hipFree(Array);
			return false;
		}
	}
	// Retrieve the result matrix
	hipMemcpy(SortedArray, Array, bytes, hipMemcpyDeviceToHost);
	// Free device memory
	hipFree(Array);
	// Success
	return true;
}

